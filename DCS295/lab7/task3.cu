#include "hip/hip_runtime.h"
#include ""
#include"matrix.h"
#include <hipDNN.h>
#include <vector>
#include <chrono>
#include <initializer_list>


template<typename T>
std::ostream& print(std::ostream& os, T* mat, int n, int c, int h, int w) {
    std::vector<T> buffer(n * c * h * w);
    hipMemcpy(buffer.data(), mat, n * c * h * w * sizeof(T), hipMemcpyDeviceToHost);
    int a = 0;
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < c; ++j) {
            os << "n = " << i << ", c = " << j << ":" << std::endl;
            print(os, &buffer[a], h, w);
            a += h * w;
        }
    }
}

template<typename T>
T* getKernelGptr(std::initializer_list<T> filter, int filt_c, int filt_h, int filt_w) {
    vector<T>kernel(filter);
    T* filt_data;
    hipMalloc(&filt_data, filt_c * filt_h * filt_w * sizeof(T));
    for (int i = 0; i < filt_c; i++)
        hipMemcpy(filt_data+ filt_h * filt_w * i, &kernel[0], sizeof(kernel[0]) * kernel.size(), hipMemcpyHostToDevice);
    return filt_data;
}
template<typename T>
T* getInputGptr(vector<T*> &input, int in_c, int in_h, int in_w) {
    T* in_data;
    hipMalloc(&in_data, in_c * in_h * in_w * sizeof(T));
    for (int i = 0; i < in_c; i++)
        hipMemcpy(in_data + in_h * in_w * i, &input[i][0], in_w * in_h * sizeof(T), hipMemcpyHostToDevice);
    return in_data;
}

int main(int argc, char* argv[]){
    if (argc != 6) {
        fprintf(stderr, "usage: TARGET [in_channels] [height] [width] [stride] [padding]\n");
        return -1;
    }
    hipdnnHandle_t cudnn;
    hipdnnCreate(&cudnn);

    int stride = atoi(argv[4]), padding = atoi(argv[5]);
    int in_n = 1,in_c = atoi(argv[1]),in_h = atoi(argv[2]),in_w = atoi(argv[3]);

    int filt_k = in_n,filt_c = in_c,filt_h = 3,filt_w = 3;

    auto filt_data = getKernelGptr<float>({ 0, 1, 0, 1, -4, 1, 0, 1, 0 }, filt_c, filt_h, filt_w);
    auto in_data = getInputGptr(getMat<float>(r, in_h, in_w, in_c), in_c, in_h, in_w);

    hipdnnTensorDescriptor_t in_desc;
    hipdnnCreateTensorDescriptor(&in_desc);
    hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, in_n, in_c, in_h, in_w);


    hipdnnFilterDescriptor_t filt_desc;
    hipdnnCreateFilterDescriptor(&filt_desc);
    hipdnnSetFilter4dDescriptor(filt_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, filt_k, filt_c, filt_h, filt_w);

    const int pad_h = padding;
    const int pad_w = padding;
    const int str_h = stride;
    const int str_w = stride;
    const int dil_h = 1;
    const int dil_w = 1;

    hipdnnConvolutionDescriptor_t conv_desc;
    hipdnnCreateConvolutionDescriptor(&conv_desc);
    hipdnnSetConvolution2dDescriptor( conv_desc, pad_h, pad_w, str_h, str_w, dil_h, dil_w, HIPDNN_CONVOLUTION, HIPDNN_DATA_FLOAT);

    int out_n;
    int out_c;
    int out_h;
    int out_w;

    hipdnnGetConvolution2dForwardOutputDim(conv_desc, in_desc, filt_desc,&out_n, &out_c, &out_h, &out_w);

    std::cout << "out_n: " << out_n << std::endl;
    std::cout << "out_c: " << out_c << std::endl;
    std::cout << "out_h: " << out_h << std::endl;
    std::cout << "out_w: " << out_w << std::endl;
    std::cout << std::endl;

    hipdnnTensorDescriptor_t out_desc;
    hipdnnCreateTensorDescriptor(&out_desc);
    hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, out_n, out_c, out_h, out_w);

    float* out_data;
    hipMalloc(&out_data, out_n * out_c * out_h * out_w * sizeof(float));

    hipdnnConvolutionFwdAlgo_t algo;
    hipdnnGetConvolutionForwardAlgorithm(cudnn, in_desc, filt_desc, conv_desc, out_desc, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 0, &algo);

    std::cout << "Convolution algorithm: " << algo << std::endl;
    std::cout << std::endl;

    size_t ws_size;
    hipdnnGetConvolutionForwardWorkspaceSize(cudnn, in_desc, filt_desc, conv_desc, out_desc, algo, &ws_size);

    float* ws_data;
    hipMalloc(&ws_data, ws_size);

    std::cout << "Workspace size: " << ws_size << std::endl;
    std::cout << std::endl;

    float alpha = 1.f;
    float beta = 0.f;
    auto timeStart = std::chrono::high_resolution_clock::now();
    hipdnnConvolutionForward(cudnn, &alpha, in_desc, in_data, filt_desc, filt_data, conv_desc, algo, ws_data, ws_size, &beta, out_desc, out_data);
    auto timeEnd = std::chrono::high_resolution_clock::now();

    auto passedTime = std::chrono::duration<double, std::milli>(timeEnd - timeStart).count();
    fprintf(stdout, "Cuda Done: %.5f (ms)\n", passedTime);



    hipFree(ws_data);
    hipFree(out_data);
    hipdnnDestroyTensorDescriptor(out_desc);
    hipdnnDestroyConvolutionDescriptor(conv_desc);
    hipFree(filt_data);
    hipdnnDestroyFilterDescriptor(filt_desc);
    hipFree(in_data);
    hipdnnDestroyTensorDescriptor(in_desc);
    hipdnnDestroy(cudnn);
    return 0;
}