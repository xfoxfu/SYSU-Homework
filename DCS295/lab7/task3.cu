#include "hip/hip_runtime.h"
#include ""
#include "errors.hpp"
#include "matrix.hpp"
#include <cassert>
#include <chrono>
#include <hipDNN.h>
#include <fmt/color.h>
#include <fmt/core.h>
#include <fmt/ostream.h>
#include <vector>

#define CUDA_GUARD(E)                                \
    {                                                \
        auto _status = E;                            \
        if (_status != hipSuccess)                  \
        {                                            \
            fmt::print(stderr,                       \
                       fg(fmt::color::red),          \
                       "Error: {}:{} ({}) {}\n",     \
                       __FILE__, __LINE__, #E,       \
                       hipGetErrorString(_status)); \
            exit(EXIT_FAILURE);                      \
        }                                            \
    }
#define CUDNN_GUARD(E)                                \
    {                                                 \
        auto _status = E;                             \
        if (_status != HIPDNN_STATUS_SUCCESS)          \
        {                                             \
            fmt::print(stderr,                        \
                       fg(fmt::color::red),           \
                       "Error: {}:{} ({}) {}\n",      \
                       __FILE__, __LINE__, #E,        \
                       hipdnnGetErrorString(_status)); \
            exit(EXIT_FAILURE);                       \
        }                                             \
    }

#define POS(m, n, p, i, j, k) \
    ((i) * (n) * (p) + (j) * (p) + (k))

Matrix conv_2d(const Matrix &in, const Matrix &ker, size_t stride, size_t bs_x, size_t bs_y)
{
    // copy device memory
    Matrix::data_t *dev_in;
    Matrix::data_t *dev_ker;
    CUDA_GUARD(hipMalloc(&dev_in, in.data_size()));
    CUDA_GUARD(hipMalloc(&dev_ker, ker.data_size()));
    CUDA_GUARD(hipMemcpy(dev_in, in._data, in.data_size(), hipMemcpyHostToDevice));
    CUDA_GUARD(hipMemcpy(dev_ker, ker._data, ker.data_size(), hipMemcpyHostToDevice));

    assert(ker.m() == ker.n());
    size_t padding = ((ker.n() - 1) / 2) * 2;

    hipdnnHandle_t cudnn;
    CUDNN_GUARD(hipdnnCreate(&cudnn));

    hipdnnTensorDescriptor_t in_desc;
    CUDNN_GUARD(hipdnnCreateTensorDescriptor(&in_desc));
    CUDNN_GUARD(hipdnnSetTensor4dDescriptor(in_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, 1, in.p(), in.m(), in.n()));

    hipdnnFilterDescriptor_t ker_desc;
    CUDNN_GUARD(hipdnnCreateFilterDescriptor(&ker_desc));
    CUDNN_GUARD(hipdnnSetFilter4dDescriptor(ker_desc, HIPDNN_DATA_DOUBLE, HIPDNN_TENSOR_NHWC, 1, ker.p(), ker.m(), ker.n()));

    size_t pad_h = padding;
    size_t pad_w = padding;
    size_t str_h = stride;
    size_t str_w = stride;
    size_t dil_h = 1;
    size_t dil_w = 1;

    hipdnnConvolutionDescriptor_t conv_desc;
    CUDNN_GUARD(hipdnnCreateConvolutionDescriptor(&conv_desc));
    CUDNN_GUARD(hipdnnSetConvolution2dDescriptor(conv_desc, pad_h, pad_w, str_h, str_w, dil_h, dil_w, HIPDNN_CONVOLUTION, HIPDNN_DATA_DOUBLE));

    int out_c;
    int out_m;
    int out_n;
    int out_p;

    CUDNN_GUARD(hipdnnGetConvolution2dForwardOutputDim(conv_desc, in_desc, ker_desc, &out_c, &out_p, &out_m, &out_n));

    hipdnnTensorDescriptor_t out_desc;
    CUDNN_GUARD(hipdnnCreateTensorDescriptor(&out_desc));
    CUDNN_GUARD(hipdnnSetTensor4dDescriptor(out_desc, HIPDNN_TENSOR_NHWC, HIPDNN_DATA_DOUBLE, out_c, out_p, out_m, out_n));

    Matrix out(out_m, out_n, out_p);
    Matrix::data_t *dev_out;
    CUDA_GUARD(hipMalloc(&dev_out, out.data_size()));

    hipdnnConvolutionFwdAlgoPerf_t perf;
    int perf_count;
    CUDNN_GUARD(cudnnGetConvolutionForwardAlgorithm_v7(
        cudnn,
        in_desc, ker_desc, conv_desc, out_desc,
        1, &perf_count, &perf));

    size_t ws_len;
    CUDNN_GUARD(hipdnnGetConvolutionForwardWorkspaceSize(
        cudnn,
        in_desc, ker_desc, conv_desc, out_desc,
        perf.algo, &ws_len));

    Matrix::data_t *ker_ws;
    CUDA_GUARD(hipMalloc(&ker_ws, ws_len));

    Matrix::data_t alpha = 1.f;
    Matrix::data_t beta = 0.f;
    CUDNN_GUARD(hipdnnConvolutionForward(
        cudnn, &alpha,
        in_desc, dev_in, ker_desc, dev_ker, conv_desc,
        perf.algo, ker_ws, ws_len,
        &beta, out_desc, dev_out));
    CUDA_GUARD(hipMemcpy(out._data, dev_out, out.data_size(), hipMemcpyDeviceToHost));

    CUDA_GUARD(hipFree(dev_in));
    CUDA_GUARD(hipFree(dev_ker));
    CUDA_GUARD(hipFree(dev_out));
    CUDA_GUARD(hipFree(ker_ws));
    CUDNN_GUARD(hipdnnDestroy(cudnn));
    CUDNN_GUARD(hipdnnDestroyConvolutionDescriptor(conv_desc));
    CUDNN_GUARD(hipdnnDestroyFilterDescriptor(ker_desc));
    CUDNN_GUARD(hipdnnDestroyTensorDescriptor(in_desc));
    CUDNN_GUARD(hipdnnDestroyTensorDescriptor(out_desc));

    return out;
}

int main(int argc, char *argv[])
{
    if (argc <= 6)
    {
        fmt::print(stderr, fg(fmt::color::red),
                   "usage: {} <height> <width> <depth> <stride> <thread.x> <thread.y> [--output]\n", argv[0]);
        return CNN_INVALID_ARGUMENTS;
    }
    size_t height = std::stoull(argv[1]);
    size_t width = std::stoull(argv[2]);
    constexpr size_t depth = 3;
    constexpr size_t filter_size = 3;
    constexpr size_t filter_count = 3;
    size_t stride = std::stoull(argv[4]);
    size_t thread_x = std::stoull(argv[5]);
    size_t thread_y = std::stoull(argv[6]);
    bool has_output = false;
    if (argc > 7 && std::strcmp(argv[7], "--output") == 0)
    {
        has_output = true;
    }

    fmt::print(fg(fmt::color::blue), "generating input\n");
    Matrix input = Matrix(height, width, depth, true);
    if (has_output)
    {
        fmt::print("{}\n", input);
    }
    fmt::print(fg(fmt::color::blue), "generating kernel\n");
    Matrix kernels[filter_count];
    for (size_t i = 0; i < filter_count; i++)
    {
        kernels[i] = Matrix(filter_size, filter_size, filter_size, true);
        if (has_output)
        {
            fmt::print("{}\n", kernels[i]);
        }
    }

    // perform convolution
    auto start = std::chrono::high_resolution_clock::now();

    fmt::print(fg(fmt::color::blue), "compute conv_2d x{}\n", filter_count);
    Matrix R[filter_count];
    for (size_t i = 0; i < filter_count; i++)
    {
        R[i] = conv_2d(input, kernels[i], stride, thread_x, thread_y);
    }

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double, std::milli> diff = end - start;
    fmt::print(fg(fmt::color::orange), "time: {} ms\n", diff.count());
    if (has_output)
    {
        for (size_t i = 0; i < filter_count; i++)
        {
            fmt::print("{}\n", R[i]);
        }
    }

    return CNN_OK;
}
